/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE
#include <stdio.h>

#include <hip/device_functions.h>

#include <stdlib.h>

#include <inttypes.h>

// #include "sm_50_atomic_functions.h"

#include <hip/hip_runtime.h>  

#define BLOCK_SIZE 1024

__global__ void histo_kernel(unsigned int* input, unsigned int* bins, unsigned int num_elements,
                             unsigned int num_bins)
{
  // initialize shared histo

  extern  __shared__ unsigned int private_histo[];

  // __shared__ unsigned int private_histo[16384];

  int tx = threadIdx.x; 

  int i = threadIdx.x + blockIdx.x * blockDim.x;

  // if (tx < BLOCK_SIZE)
  //     for(unsigned int j = 0; j < num_bins; ++j) {
  //      private_histo[j] = 0;
  //     }
  // }
  for (int p = 0; p < (BLOCK_SIZE+num_bins-1)/BLOCK_SIZE; p++) {
    if ( p * tx < num_bins){
      private_histo[ p*tx] = 0;
    }
  }

    // if (tx < num_bins) private_histo[tx] = 0;

  __syncthreads();

  // stride is total number of threads
  int stride = blockDim.x * gridDim.x;

  // All threads handle blockDim.x * gridDim.x
  // consecutive elements
  while (i < num_elements) {
    atomicAdd( &(private_histo[input[i]]), 1);
    i += stride;
  }

        // wait for all other threads in the block to finish
        __syncthreads();

  for (int q = 0; q < (BLOCK_SIZE+num_bins-1)/BLOCK_SIZE; q++) {
    if ( q * BLOCK_SIZE + tx < num_bins &&  threadIdx.x < BLOCK_SIZE) {
        atomicAdd(&(bins[q*BLOCK_SIZE+threadIdx.x]), private_histo[q*BLOCK_SIZE+threadIdx.x] );
    }
  }


      // if (threadIdx.x < BLOCK_SIZE) {
      //   atomicAdd(&(bins[threadIdx.x]), private_histo[threadIdx.x] );
      // }
}



/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
void histogram(unsigned int* input, unsigned int* bins, unsigned int num_elements,
        unsigned int num_bins) {

    // INSERT CODE HERE

    // dim3 dim_grid((num_elements+BLOCK_SIZE-1)/BLOCK_SIZE);

  // const unsigned int num_bins;

  dim3 dim_grid(16,1);

  // dim3 dim_grid(BLOCK_SIZE,1);

  dim3 dim_block(BLOCK_SIZE,1);

	histo_kernel<<<dim_grid, dim_block, num_bins * sizeof(unsigned int)>>>( input, bins, num_elements, num_bins);

}


