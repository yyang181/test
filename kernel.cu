/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

    
__global__ void VecAdd(int n, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (1 * n) vector
     *   where B is a (1 * n) vector
     *   where C is a (1 * n) vector
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE

	int i = blockIdx.x*blockDim.x + threadIdx.x;

	 C[i] = A[i] + B[i];
}


void basicVecAdd( float *A,  float *B, float *C, int n)
{

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = 256; 

    //INSERT CODE HERE
        dim3 dim_grid, dim_block;
	dim_grid = (n + BLOCK_SIZE -1)/BLOCK_SIZE;  
	dim_block = 256 ;
	VecAdd<<<dim_grid, dim_block>>>(n, A, B, C);

}
